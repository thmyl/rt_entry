#include "hip/hip_runtime.h"
#include "graph.h"
#include <immintrin.h>
#include <omp.h>

__global__ void check_entries_kernel(uint *d_entries, uint n_entries, uint nq, uint *d_gt, uint gt_k, float *d_recall_1, float *d_recall_10, float *d_recall_100){
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < nq){
    uint offset = tid * n_entries;
    for(int i=0; i<gt_k; i++){
      uint gt = d_gt[tid * gt_k + i];
      for(int j=0; j<n_entries; j++){
        if(gt == d_entries[offset + j]){
          if(i<1)d_recall_1[tid] += 1;
          if(i<10)d_recall_10[tid] += 1;
          if(i<100)d_recall_100[tid] += 1;
          break;
        }
      }
    }
  }
}

void Graph::check_entries(thrust::device_vector<uint> &d_gt_){
  printf("checking entries...\n");

  auto *d_entries_ptr = thrust::raw_pointer_cast(d_entries.data());
  auto *d_gt_ptr = thrust::raw_pointer_cast(d_gt_.data());
  thrust::device_vector<float> d_recall_1(nq, 0);
  thrust::device_vector<float> d_recall_10(nq, 0);
  thrust::device_vector<float> d_recall_100(nq, 0);
  check_entries_kernel<<<(nq + 255)/256, 256>>>(d_entries_ptr, n_entries, nq, d_gt_ptr, gt_k,
                                                    thrust::raw_pointer_cast(d_recall_1.data()),
                                                    thrust::raw_pointer_cast(d_recall_10.data()),
                                                    thrust::raw_pointer_cast(d_recall_100.data()));
  CUDA_SYNC_CHECK();
  thrust::host_vector<float> h_recall_1 = d_recall_1;
  thrust::host_vector<float> h_recall_10 = d_recall_10;
  thrust::host_vector<float> h_recall_100 = d_recall_100;
  float sum_1 = 0;
  float sum_10 = 0;
  float sum_100 = 0;
  for(int i=0; i<nq; i++){
    sum_1 += h_recall_1[i];
    sum_10 += h_recall_10[i];
    sum_100 += h_recall_100[i];
  }
  sum_1 = sum_1 / nq;
  sum_10 = sum_10 / nq / 10;
  sum_100 = sum_100 / nq / 100;
  printf("recall@1 = %f\n", sum_1);
  printf("recall@10 = %f\n", sum_10);
  printf("recall@100 = %f\n", sum_100);

  std::ofstream outfile;
  outfile.open(OUTFILE, std::ios_base::app);
  outfile << "entries recall:\n";
  outfile <<  "recall@1 = " << sum_1 << " ms\n";
  outfile <<  "recall@10 = " << sum_10 << " ms\n";
  outfile <<  "recall@100 = " << sum_100 << " ms\n" << std::flush;
  outfile.close();
}


__global__ void check_results_kernel(uint *d_results, uint n_results, uint nq, uint *d_gt, uint gt_k, float *d_recall_1, float *d_recall_10, float *d_recall_100){
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < nq){
    uint offset = tid * n_results;
    for(int i=0; i<gt_k; i++){
      uint gt = d_gt[tid * gt_k + i];
      for(int j=0; j<n_results; j++){
        if(gt == d_results[offset + j]){
          if(i<1)d_recall_1[tid] += 1;
          if(i<10)d_recall_10[tid] += 1;
          if(i<100)d_recall_100[tid] += 1;
          break;
        }
      }
    }
  }
}

void Graph::check_results(thrust::device_vector<uint> &d_gt_){
  printf("checking results...\n");

  auto *d_results_ptr = thrust::raw_pointer_cast(d_results.data());
  auto *d_gt_ptr = thrust::raw_pointer_cast(d_gt_.data());
  printf("gt_k = %d\n", gt_k);
  thrust::device_vector<float> d_recall_1(nq, 0);
  thrust::device_vector<float> d_recall_10(nq, 0);
  thrust::device_vector<float> d_recall_100(nq, 0);
  check_results_kernel<<<(nq + 255)/256, 256>>>(d_results_ptr, topk, nq, d_gt_ptr, gt_k,
                                                    thrust::raw_pointer_cast(d_recall_1.data()),
                                                    thrust::raw_pointer_cast(d_recall_10.data()),
                                                    thrust::raw_pointer_cast(d_recall_100.data()));
  CUDA_SYNC_CHECK();
  thrust::host_vector<float> h_recall_1 = d_recall_1;
  thrust::host_vector<float> h_recall_10 = d_recall_10;
  thrust::host_vector<float> h_recall_100 = d_recall_100;
  float sum_1 = 0;
  float sum_10 = 0;
  float sum_100 = 0;
  for(int i=0; i<nq; i++){
    sum_1 += h_recall_1[i];
    sum_10 += h_recall_10[i];
    sum_100 += h_recall_100[i];
  }
  sum_1 = sum_1 / nq;
  sum_10 = sum_10 / nq / 10;
  sum_100 = sum_100 / nq / 100;
  printf("recall@1 = %f\n", sum_1);
  printf("recall@10 = %f\n", sum_10);
  printf("recall@100 = %f\n", sum_100);

  std::ofstream outfile;
  outfile.open(OUTFILE, std::ios_base::app);
  // outfile << "results recall:\n";
  // outfile <<  "recall@1 = " << sum_1 << " ms\n";
  outfile <<  "recall@10 = " << sum_10 << " ms\n" << std::flush;
  // outfile <<  "recall@100 = " << sum_100 << " ms\n" << std::flush;
  outfile.close();
}

bool cmp(Pair a, Pair b){
  return a.dist < b.dist;
}

/*void Graph::paralled_reorder(uint* candidates, uint* results, uint n_candidates, uint topk, uint dim_, uint nq, float* points, uint np, float* queries, Pair* candidates_dist){
  for(int q_id=0; q_id<nq; q_id++){
    Pair* cur_candidates_dist = candidates_dist + q_id * n_candidates;
    for(int j=0; j<n_candidates; j++){
      uint p_id = candidates[q_id*n_candidates+j];
      cur_candidates_dist[j].id = p_id;
      float dis = 0;
      for(int d=0; d<dim_; d++){
        dis = dis + (points[p_id*dim_+d] - queries[q_id*dim_+d]) * (points[p_id*dim_+d] - queries[q_id*dim_+d]);
      }
      cur_candidates_dist[j].dist = dis;
    }
    std::sort(cur_candidates_dist, cur_candidates_dist+n_candidates, cmp);
    for(int j=0; j<topk; j++){
      results[q_id*topk+j] = cur_candidates_dist[j].id;
    }
  }
}*/

void Graph::parallel_reorder(uint* candidates, uint* results, uint n_candidates, uint topk, uint dim_, uint nq, float* points, uint np, float* queries, Pair* candidates_dist) {
  // int max_threads = omp_get_max_threads();
  int max_threads = 64;

  #pragma omp parallel for schedule(dynamic) num_threads(max_threads)
  for (int q_id_ = 0; q_id_ < nq; ++q_id_) {
    uint q_id = q_id_;
    Pair* cur_candidates_dist = candidates_dist + q_id * n_candidates;

    // 计算距离并填充cur_candidates_dist
    // for (int j = 0; j < n_candidates; ++j) {
    //   uint p_id = candidates[q_id * n_candidates + j];
    //   cur_candidates_dist[j].id = p_id;
    //   float dis = 0.0f;
    //   for (int d = 0; d < dim_; ++d) {
    //     float diff = points[p_id * dim_ + d] - queries[q_id * dim_ + d];
    //     dis += diff * diff;
    //   }
    //   cur_candidates_dist[j].dist = dis;
    // }

    // 计算距离并填充cur_candidates_dist
    for (int j = 0; j < n_candidates; ++j) {
      long long p_id = candidates[q_id * n_candidates + j];
      // p_id = j;
      cur_candidates_dist[j].id = p_id;
      __m256 dis_vec = _mm256_setzero_ps(); // 初始化为0

      // 使用SIMD计算距离
      for (int d = 0; d < dim_; d += 8) {
        // 加载points和queries的数据到向量寄存器
        __m256 point_vec = _mm256_loadu_ps(&points[p_id * dim_ + d]);
        __m256 query_vec = _mm256_loadu_ps(&queries[q_id * dim_ + d]);

        // 计算差值
        __m256 diff_vec = _mm256_sub_ps(point_vec, query_vec);

        // 计算平方
        __m256 square_vec = _mm256_mul_ps(diff_vec, diff_vec);

        // 累加到dis_vec
        dis_vec = _mm256_add_ps(dis_vec, square_vec);
      }

      // 将向量结果汇总为标量
      float dis = 0.0f;
      float temp_dis[8];
      _mm256_storeu_ps(temp_dis, dis_vec); // 存储向量结果到数组
      for (int k = 0; k < 8 && (k + 8 * ((dim_ - 1) / 8)) < dim_; ++k) {
        dis += temp_dis[k];
      }

      // 处理剩余的维度（如果dim_不是8的倍数）
      for (int d = (dim_ / 8) * 8; d < dim_; ++d) {
        float diff = points[p_id * dim_ + d] - queries[q_id * dim_ + d];
        dis += diff * diff;
      }

      cur_candidates_dist[j].dist = dis;
    }

    std::sort(cur_candidates_dist, cur_candidates_dist + n_candidates, cmp);

    for (int j = 0; j < topk; ++j) {
      results[q_id * topk + j] = cur_candidates_dist[j].id;
    }
  }
}

void Graph::CopyHostToDevice(thrust::host_vector<float> &h_data, thrust::device_vector<float> &d_data, uint n, uint d, uint d_){
  d_data.resize(n*d_);
  for(int i=0; i<n; i++){
    thrust::copy(h_data.begin() + i*d, h_data.begin()+ i*d + d_, d_data.begin() + i*d_);
  }
}

__global__ void calc_dist(float* data1, float* data2, uint n, uint d, uint* belong, float* dist, int cluster_id){
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < n){
    float dis = 0;
    for(int i=0; i<d; i++){
      dis += (data1[tid*d+i] - data2[tid*d+i]) * (data1[tid*d+i] - data2[tid*d+i]);
    }
    if(dis < dist[tid]){
      dist[tid] = dis;
      belong[tid] = cluster_id;
    }
    // float old_dist = dist[tid];
    // if (dis < old_dist) {
    //   dist[tid] = dis;
    //   belong[tid] = cluster_id;
    // }
    // dist[tid] = dis;
  }
}

void Graph::NearestCluster(){
  printf("NearestCluster...\n");
  uint* d_belong_ptr = thrust::raw_pointer_cast(d_belong.data());
  float* d_dist_ptr = thrust::raw_pointer_cast(d_dist.data());

  for(int cluster_id=0; cluster_id<n_clusters; cluster_id++){
    matrixMultiply(handle_, d_queries_, kpca->d_rotation[cluster_id], kpca->d_transforms[cluster_id], nq, kpca->n_components, dim_, 1.0, -1.0);
    matrixMultiply(handle_, kpca->d_transforms[cluster_id], kpca->d_t[cluster_id], kpca->d_reconstructed[cluster_id], nq, dim_, kpca->n_components, 1.0, 1.0);
    // printf("d_query_size = %d\n", d_queries_.size());
    // printf("d_rotation_size = %d\n", kpca->d_rotation[cluster_id].size());
    // printf("d_transforms_size = %d\n", kpca->d_transforms[cluster_id].size());
    // printf("d_t_size = %d\n", kpca->d_t[cluster_id].size());
    // printf("d_reconstructed_size = %d\n", kpca->d_reconstructed[cluster_id].size());
    // printf("d_belong_size = %d\n", d_belong.size());
    // printf("d_dist_size = %d\n", d_dist.size());
    float* d_queries_ptr = thrust::raw_pointer_cast(d_queries_.data());
    float* d_reconstructed_ptr = thrust::raw_pointer_cast(kpca->d_reconstructed[cluster_id].data());
    calc_dist<<<(nq + 255)/256, 256>>>(d_queries_ptr, d_reconstructed_ptr, nq, dim_, d_belong_ptr, d_dist_ptr, cluster_id);
    // hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "CUDA kernel error in cluster %d: %s\n", cluster_id, hipGetErrorString(err));
      return;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      fprintf(stderr, "CUDA synchronization error in cluster %d: %s\n", cluster_id, hipGetErrorString(err));
      return;
    }
  }

  // printf("d_size = %d\n", d_belong.size());
  // printf("h_size = %d\n", h_belong.size());
  // thrust::copy(d_belong.begin(), d_belong.end(), h_belong.begin());
  // for(int i=0; i<100; i++)printf("%d ", h_belong[i]);printf("\n");
}